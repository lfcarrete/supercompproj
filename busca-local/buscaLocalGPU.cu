#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>  
#include <iomanip>
#include <fstream>
#include <string>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <random>
#include <algorithm>

using namespace std;

struct ponto
{
    int id;
    double x;
    double y;
};

double calcDist(ponto a, ponto b) {
    return sqrt(pow(b.x - a.x, 2) + pow(b.y - a.y, 2));
}

struct saxpy {
    int *id;
    double *todasDists;
    int numCity;
    saxpy(int *id_, double *todasDists_, int numCity_) : id(id_), todasDists(todasDists_), numCity(numCity_) {};

    __host__ __device__
    int operator()(const int x) {
///////////// Dist inicial e preenche temp
        int index = x*numCity;
        double distTot = 0;
        int *temp = new int[numCity];
        int id_a;
        int id_b;

        for(int i = 0; i < numCity; i ++){
            if(i == numCity-1){
                id_a = id[index + numCity-1];
                id_b = id[index];
            } else {
                id_a = id[index + i];
                id_b = id[index + i + 1];
            }
            distTot += todasDists[id_a*numCity + id_b];
            
            temp[i] = id_a;
        }
///////////////////////



        bool madeChange = true;
        int *tempReset = new int[numCity];
        for(int v = 0; v < numCity; v++){
            tempReset[v] = temp[v];
        }

        int tempVal = 0;
        double distTemp = 0;
        while (madeChange){
            madeChange = false;
            for(int i = 0; i < numCity; i++){
                if(i == numCity-1){
                    tempVal = temp[i];
                    temp[i] = temp[0];
                    temp[0] = tempVal;
                } else {
                    tempVal   = temp[i];
                    temp[i]   = temp[i+1];
                    temp[i+1] = tempVal;
                }

                for(int c = 0; c < numCity; c ++){
                    if(c == numCity-1){
                        id_a = temp[c];
                        id_b = temp[0];
                    } else {
                        id_a = temp[c];
                        id_b = temp[c + 1];
                    }
                    distTemp += todasDists[id_a*numCity + id_b];
                }

                if(distTemp < distTot){
                    madeChange = true;
                    
                    distTot = distTemp;

                    for(int e = 0; e < numCity; e ++){
                        id[index + e] = temp[e];
                        tempReset[e] = temp[e];
                    }
                } else {
                    for(int v = 0; v < numCity; v++){
                        temp[v] = tempReset[v];
                    }
                }
                distTemp = 0;
            }
        }

        return distTot;
    }
};

vector<ponto> read_file(int numLines) {
    vector<ponto> pontos;
    string line;
    for(int i = 0; i < numLines; i++) {
        ponto novoItem;
        novoItem.id = i;
        cin >> line;
        novoItem.x = stof(line);
        cin >> line;
        novoItem.y = stof(line);
        pontos.push_back(novoItem);
    }
    return pontos;
}


int main(){
    int numLines;
    int seed = 10;
    cin >> numLines;

    vector<ponto> pontos = read_file(numLines);

    thrust::host_vector<double> todasDist(numLines*numLines);

    for(int j = 0; j < numLines; j++){
        for(int i = 0; i < numLines; i++){
            int index = (i * numLines) + j;
            todasDist[index] = calcDist(pontos[j], pontos[i]);
            //cout << todasDist[index];
        }
    }

    thrust::device_vector<double> todasDistGPU(todasDist);    
    default_random_engine generator(seed);
    
    thrust::host_vector<int> todasPerm(10*numLines*numLines);
    vector<ponto> permut;
    for(int rounds = 0; rounds < 10*numLines; rounds ++){
        permut = pontos;
        shuffle(permut.begin(), permut.end(), generator);

        for(int i = 0; i < numLines; i++){
            int index = (rounds * numLines) + i;
            todasPerm[index] = permut[i].id;
        }
    }

    thrust::device_vector<int> todasPermGPU(todasPerm);    

    auto iter = thrust::make_counting_iterator(0);

    thrust::device_vector<int> result(10*numLines);

    thrust::transform(
        iter,
        iter + 10*numLines, 
        result.begin(), 
        saxpy(thrust::raw_pointer_cast(todasPermGPU.data()), thrust::raw_pointer_cast(todasDistGPU.data()), numLines)
    ); 
    
    cout << result.size() << endl;
    for(int rounds = 0; rounds < 10*numLines; rounds ++){
        for(int i = 0; i < numLines; i++){
            int index = (rounds * numLines) + i;
            cout << todasPermGPU[index] << " ";
        }
        cout << " : "<< result[rounds];
        cout << endl;
    }
    return 0; 
}

